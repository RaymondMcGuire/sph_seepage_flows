#include "hip/hip_runtime.h"
/*** 
 * @Author: Xu.WANG raymondmgwx@gmail.com
 * @Date: 2023-03-22 15:38:55
 * @LastEditors: Xu.WANG raymondmgwx@gmail.com
 * @LastEditTime: 2023-03-23 15:58:54
 * @FilePath: \sph_seepage_flows\seepage_flow_cuda\src\kiri_pbs_cuda\solver\seepage_flow\cuda_dfsph_sf_solver.cu
 * @Description: 
 * @Copyright (c) 2023 by Xu.WANG, All Rights Reserved. 
 */
#include <kiri_pbs_cuda/solver/seepageflow/cuda_dfsph_sf_solver.cuh>
#include <kiri_pbs_cuda/solver/seepageflow/cuda_dfsph_sf_solver_gpu.cuh>
#include <kiri_pbs_cuda/thrust_helper/helper_thrust.cuh>
#include <thrust/device_ptr.h>
namespace KIRI {

void CudaDFSphSFSolver::ComputeDensity(
    CudaSFParticlesPtr &particles, CudaBoundaryParticlesPtr &boundaries,
    const float rho0, const float rho1, const CudaArray<size_t> &cellStart,
    const CudaArray<size_t> &boundaryCellStart, const float3 lowestPoint,
    const float kernelRadius, const int3 gridSize) {

  auto data = std::dynamic_pointer_cast<CudaDFSFParticles>(particles);
  _ComputeSFDensity_CUDA<<<mCudaGridSize, KIRI_CUBLOCKSIZE>>>(
      data->GetDensityPtr(), data->GetLabelPtr(),
      data->GetPosPtr(), data->GetMassPtr(), rho0, rho1,
      data->Size(), cellStart.Data(), boundaries->GetPosPtr(),
      boundaries->GetVolumePtr(), boundaryCellStart.Data(), gridSize,
      ThrustHelper::Pos2GridXYZ<float3>(lowestPoint, kernelRadius, gridSize),
      ThrustHelper::GridXYZ2GridHash(gridSize), CubicKernel(kernelRadius));
  KIRI_CUCALL(hipDeviceSynchronize());
  KIRI_CUKERNAL();
}

void CudaDFSphSFSolver::AdvectDFSPHVelocity(CudaDFSFParticles &fluids) {
  
  fluids->AdvectFluidVel(mDt);
}

void CudaDFSphSFSolver::ComputeTimeStepsByCFL(CudaDFSFParticles &fluids,
                                            const float particleRadius,
                                            const float timeIntervalInSeconds) {

  
  _ComputeVelMag_CUDA<<<mCudaGridSize, KIRI_CUBLOCKSIZE>>>(
      fluids->GetVelMagPtr(),fluids->GetLabelPtr(), fluids->GetVelPtr(), fluids->GetAccPtr(), mDt, fluids->Size());

  auto vel_mag_array = thrust::device_pointer_cast(fluids->GetVelMagPtr());
  float max_vel_mag =
      *(thrust::max_element(vel_mag_array, vel_mag_array + fluids->Size()));

  auto diam = 2.f * particleRadius;
  mDt = CFL_FACTOR * 0.4f * (diam / sqrt(max_vel_mag));
  mDt = max(mDt, CFL_MIN_TIMESTEP_SIZE);
  mDt = min(mDt, CFL_MAX_TIMESTEP_SIZE);

  mNumOfSubTimeSteps = static_cast<int>(std::ceil(timeIntervalInSeconds / mDt));

  KIRI_CUCALL(hipDeviceSynchronize());
  KIRI_CUKERNAL();
}

void CudaDFSphSFSolver::ComputeDFSPHAlpha(
    CudaDFSFParticles &fluids, CudaBoundaryParticlesPtr &boundaries,
    const float rho0, const CudaArray<size_t> &cellStart,
    const CudaArray<size_t> &boundaryCellStart, const float3 lowestPoint,
    const float kernelRadius, const int3 gridSize) {
  
  _ComputeAlpha_CUDA<<<mCudaGridSize, KIRI_CUBLOCKSIZE>>>(
      fluids->GetAlphaPtr(),fluids->GetLabelPtr(), fluids->GetPosPtr(), fluids->GetMassPtr(), fluids->GetDensityPtr(),
      rho0, fluids->Size(), cellStart.Data(), boundaries->GetPosPtr(),
      boundaries->GetVolumePtr(), boundaryCellStart.Data(), gridSize,
      ThrustHelper::Pos2GridXYZ<float3>(lowestPoint, kernelRadius, gridSize),
      ThrustHelper::GridXYZ2GridHash(gridSize), CubicKernelGrad(kernelRadius));

  KIRI_CUCALL(hipDeviceSynchronize());
  KIRI_CUKERNAL();
}

size_t CudaDFSphSFSolver::ApplyDivergenceSolver(
    CudaDFSFParticles &fluids, CudaBoundaryParticlesPtr &boundaries,
    const float rho0, const CudaArray<size_t> &cellStart,
    const CudaArray<size_t> &boundaryCellStart, const float3 lowestPoint,
    const float kernelRadius, const int3 gridSize) {
  
  auto num = fluids->Size();

  // Compute velocity of density change
  _ComputeDivgenceError_CUDA<<<mCudaGridSize, KIRI_CUBLOCKSIZE>>>(
      fluids->GetStiffPtr(), fluids->GetDensityErrorPtr(),fluids->GetLabelPtr(),  fluids->GetAlphaPtr(),
      fluids->GetPosPtr(), fluids->GetVelPtr(), fluids->GetMassPtr(), fluids->GetDensityPtr(), rho0,
      mDt, num, cellStart.Data(), boundaries->GetPosPtr(), boundaries->GetVolumePtr(),
      boundaryCellStart.Data(), gridSize,
      ThrustHelper::Pos2GridXYZ<float3>(lowestPoint, kernelRadius, gridSize),
      ThrustHelper::GridXYZ2GridHash(gridSize), CubicKernelGrad(kernelRadius));

  auto iter = 0;
  auto total_error = std::numeric_limits<float>::max();

  while ((total_error > mDivergenceErrorThreshold * num * rho0 ||
          (iter < mDivergenceMinIter)) &&
         (iter < mDivergenceMaxIter)) {

    _CorrectDivergenceByJacobi_CUDA<<<mCudaGridSize, KIRI_CUBLOCKSIZE>>>(
        fluids->GetVelPtr(), fluids->GetStiffPtr(),fluids->GetLabelPtr(),  fluids->GetPosPtr(), fluids->GetMassPtr(), rho0,
        num, cellStart.Data(), boundaries->GetPosPtr(), boundaries->GetVolumePtr(),
        boundaryCellStart.Data(), gridSize,
        ThrustHelper::Pos2GridXYZ<float3>(lowestPoint, kernelRadius, gridSize),
        ThrustHelper::GridXYZ2GridHash(gridSize),
        CubicKernelGrad(kernelRadius));

    _ComputeDivgenceError_CUDA<<<mCudaGridSize, KIRI_CUBLOCKSIZE>>>(
        fluids->GetStiffPtr(), fluids->GetDensityErrorPtr(), fluids->GetAlphaPtr(),
        fluids->GetPosPtr(), fluids->GetVelPtr(), fluids->GetMassPtr(), fluids->GetDensityPtr(),
        rho0, mDt, num, cellStart.Data(), boundaries->GetPosPtr(),
        boundaries->GetVolumePtr(), boundaryCellStart.Data(), gridSize,
        ThrustHelper::Pos2GridXYZ<float3>(lowestPoint, kernelRadius, gridSize),
        ThrustHelper::GridXYZ2GridHash(gridSize),
        CubicKernelGrad(kernelRadius));

    iter++;

    total_error =
        thrust::reduce(thrust::device_ptr<float>(fluids->GetDensityErrorPtr()),
                       thrust::device_ptr<float>(fluids->GetDensityErrorPtr() + num),
                       0.f, ThrustHelper::AbsPlus<float>());
  }

  // printf("divergence iter=%d, total_error=%.6f \n", iter,
  //        total_error);
  KIRI_CUCALL(hipDeviceSynchronize());
  KIRI_CUKERNAL();

  return iter;
}

size_t CudaDFSphSFSolver::ApplyPressureSolver(
    CudaDFSFParticles &fluids, CudaBoundaryParticlesPtr &boundaries,
    const float rho0, const CudaArray<size_t> &cellStart,
    const CudaArray<size_t> &boundaryCellStart, const float3 lowestPoint,
    const float kernelRadius, const int3 gridSize) {
  
  auto num = fluids->Size();

  // use warm stiff
  _CorrectPressureByJacobi_CUDA<<<mCudaGridSize, KIRI_CUBLOCKSIZE>>>(
      fluids->GetVelPtr(), fluids->GetWarmStiffPtr(),fluids->GetLabelPtr(),  fluids->GetPosPtr(), fluids->GetMassPtr(),
      rho0, mDt, num, cellStart.Data(), boundaries->GetPosPtr(),
      boundaries->GetVolumePtr(), boundaryCellStart.Data(), gridSize,
      ThrustHelper::Pos2GridXYZ<float3>(lowestPoint, kernelRadius, gridSize),
      ThrustHelper::GridXYZ2GridHash(gridSize), CubicKernelGrad(kernelRadius));

  _ComputeDensityError_CUDA<<<mCudaGridSize, KIRI_CUBLOCKSIZE>>>(
      fluids->GetDensityErrorPtr(), fluids->GetStiffPtr(), fluids->GetLabelPtr(), fluids->GetAlphaPtr(),
      fluids->GetPosPtr(), fluids->GetVelPtr(), fluids->GetMassPtr(), fluids->GetDensityPtr(), rho0,
      mDt, num, cellStart.Data(), boundaries->GetPosPtr(), boundaries->GetVolumePtr(),
      boundaryCellStart.Data(), gridSize,
      ThrustHelper::Pos2GridXYZ<float3>(lowestPoint, kernelRadius, gridSize),
      ThrustHelper::GridXYZ2GridHash(gridSize), CubicKernelGrad(kernelRadius));

  // reset warm stiffness
  KIRI_CUCALL(hipMemcpy(fluids->GetWarmStiffPtr(), fluids->GetStiffPtr(),
                         sizeof(float) * num, hipMemcpyDeviceToDevice));

  auto iter = 0;
  auto total_error = std::numeric_limits<float>::max();

  while ((total_error > mPressureErrorThreshold * num * rho0 ||
          (iter < mPressureMinIter)) &&
         (iter < mPressureMaxIter)) {

    _CorrectPressureByJacobi_CUDA<<<mCudaGridSize, KIRI_CUBLOCKSIZE>>>(
        fluids->GetVelPtr(),fluids->GetLabelPtr(),  fluids->GetStiffPtr(), fluids->GetPosPtr(), fluids->GetMassPtr(), rho0,
        mDt, num, cellStart.Data(), boundaries->GetPosPtr(),
        boundaries->GetVolumePtr(), boundaryCellStart.Data(), gridSize,
        ThrustHelper::Pos2GridXYZ<float3>(lowestPoint, kernelRadius, gridSize),
        ThrustHelper::GridXYZ2GridHash(gridSize),
        CubicKernelGrad(kernelRadius));

    _ComputeDensityError_CUDA<<<mCudaGridSize, KIRI_CUBLOCKSIZE>>>(
        fluids->GetDensityErrorPtr(), fluids->GetStiffPtr(),fluids->GetLabelPtr(),  fluids->GetAlphaPtr(),
        fluids->GetPosPtr(), fluids->GetVelPtr(), fluids->GetMassPtr(), fluids->GetDensityPtr(),
        rho0, mDt, num, cellStart.Data(), boundaries->GetPosPtr(),
        boundaries->GetVolumePtr(), boundaryCellStart.Data(), gridSize,
        ThrustHelper::Pos2GridXYZ<float3>(lowestPoint, kernelRadius, gridSize),
        ThrustHelper::GridXYZ2GridHash(gridSize),
        CubicKernelGrad(kernelRadius));

    thrust::transform(thrust::device, fluids->GetWarmStiffPtr(),
                      fluids->GetWarmStiffPtr() + num, fluids->GetStiffPtr(),
                      fluids->GetWarmStiffPtr(), thrust::plus<float>());
    iter++;

    if (iter >= mPressureMinIter) {
      total_error = thrust::reduce(
          thrust::device_ptr<float>(fluids->GetDensityErrorPtr()),
          thrust::device_ptr<float>(fluids->GetDensityErrorPtr() + num), 0.f,
          ThrustHelper::AbsPlus<float>());
    }
  }

  //   printf("Total Iteration Num=%d; Total Error=%.6f; Threshold=%.6f \n",
  //   iter,
  //          total_error, mPressureErrorThreshold * num * rho0);

  KIRI_CUCALL(hipDeviceSynchronize());
  KIRI_CUKERNAL();

  return iter;
}


} // namespace KIRI
