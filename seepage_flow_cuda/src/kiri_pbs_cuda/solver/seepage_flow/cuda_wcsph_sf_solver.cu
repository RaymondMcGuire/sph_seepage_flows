#include "hip/hip_runtime.h"
/*
 * @Author: Xu.WANG
 * @Date: 2021-02-03 17:49:11
 * @LastEditTime: 2021-04-08 00:05:31
 * @LastEditors: Xu.WANG
 * @Description:
 * @FilePath:
 * \Kiri\KiriPBSCuda\src\kiri_pbs_cuda\solver\seepage_flow\cuda_wcsph_sf_solver.cu
 */

#include <kiri_pbs_cuda/solver/seepageflow/cuda_wcsph_sf_solver.cuh>
#include <kiri_pbs_cuda/solver/seepageflow/cuda_wcsph_sf_solver_gpu.cuh>
#include <kiri_pbs_cuda/thrust_helper/helper_thrust.cuh>
#include <thrust/device_ptr.h>
namespace KIRI {

void CudaWCSphSFSolver::ComputePressure(CudaSFParticlesPtr &particles,
                                        const float rho0, const float stiff) {
  _ComputeSFPressureByTait_CUDA<<<mCudaGridSize, KIRI_CUBLOCKSIZE>>>(
      particles->GetLabelPtr(), particles->GetDensityPtr(),
      particles->GetPressurePtr(), particles->Size(), rho0, stiff,
      mNegativeScale);

  KIRI_CUCALL(hipDeviceSynchronize());
  KIRI_CUKERNAL();
}

void CudaWCSphSFSolver::ComputeSubTimeStepsByCFL(CudaSFParticlesPtr &particles,
                                                 const float sphMass,
                                                 const float dt,
                                                 const float kernelRadius,
                                                 float renderInterval) {

  auto accArray = thrust::device_pointer_cast(particles->GetAccPtr());
  float3 maxAcc =
      *(thrust::max_element(accArray, accArray + particles->Size(),
                            ThrustHelper::CompareLengthCuda<float3>()));

  float maxForceMagnitude = length(maxAcc) * sphMass;
  float timeStepLimitBySpeed =
      mTimeStepLimitBySpeedFactor * kernelRadius / mSpeedOfSound;
  float timeStepLimitByForce =
      mTimeStepLimitByForceFactor *
      std::sqrt(kernelRadius * sphMass / maxForceMagnitude);
  float desiredTimeStep =
      std::min(mTimeStepLimitScale *
                   std::min(timeStepLimitBySpeed, timeStepLimitByForce),
               dt);

  mNumOfSubTimeSteps = static_cast<size_t>(renderInterval / desiredTimeStep);

  KIRI_CUCALL(hipDeviceSynchronize());
  KIRI_CUKERNAL();
}

} // namespace KIRI
