/**
 * @ Author: Justin Francis
 * @ Url: https://wagonhelm.github.io/articles/2018-03/detecting-cuda-capability-with-cmake
 * @ Create Time: 2020-07-04 15:04:53
 * @ Modified by: Xu Wang
 * @ Modified time: 2020-07-04 15:40:12
 * @ Description:
 */
#pragma once
#include <hip/hip_runtime.h>

#pragma warning(disable: 4819)

#include <stdio.h>

int main(int argc, char **argv){
    hipDeviceProp_t dP;
    float min_cc = 3.0;

    int rc = hipGetDeviceProperties(&dP, 0);
    if(rc != hipSuccess) {
        hipError_t error = hipGetLastError();
        printf("CUDA error: %s", hipGetErrorString(error));
        return rc; /* Failure */
    }
    if((dP.major+(dP.minor/10)) < min_cc) {
        printf("Min Compute Capability of %2.1f required:  %d.%d found\n Not Building CUDA Code", min_cc, dP.major, dP.minor);
        return 1; /* Failure */
    } else {
        //printf("-arch=sm_%d%d", dP.major, dP.minor);
        printf("%d%d", dP.major, dP.minor);
        return 0; /* Success */
    }
}